#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <fstream>

#include "SearchKernel.cuh"
using namespace std;

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }


typedef struct __align__(16)  {
	int startPosition;
	int hitsNum;
	double z;
} sensorInfo;

struct track {
	double m_x0;
	double m_tx;
	double m_y0;
	double m_ty;

	double m_s0;
	double m_sx;
	double m_sz;
	double m_sxz;
	double m_sz2;

	double m_u0;
	double m_uy;
	double m_uz;
	double m_uyz;
	double m_uz2;

	int internalId;
	int trackHitsNum; // hmm
	int firstHit; 	 //	hmm

	bool m_backward;
};

__device__ __constant__ double 	m_maxXSlope			= 0.400;
__device__ __constant__ double 	m_maxYSlope			= 0.300;
__device__ __constant__ double 	m_maxZForRBeamCut	= 200.0;
__device__ __constant__ double 	m_maxR2Beam			= 1.0 ;
__device__ __constant__ int 	m_maxMissed			= 4;
__device__ __constant__ double 	m_extraTol			= 0.150;
__device__ __constant__ double 	m_maxChi2ToAdd		= 100.0;
__device__ __constant__ double 	m_maxChi2SameSensor	= 16.0;
__device__ __constant__ double  m_maxChi2Short		= 6.0 ;
__device__ __constant__ double  m_maxChi2PerHit		= 16.0;
__device__ __constant__ int 	m_sensNum			= 48;


__device__ double zBeam(track *tr) {
	return -( tr->m_x0 * tr->m_tx + tr->m_y0 * tr->m_ty ) / ( tr->m_tx * tr->m_tx + tr->m_ty * tr->m_ty );
}

__device__ double r2AtZ( double z , track *tr) {
    double xx = tr->m_x0 + z * tr->m_tx;
    double yy = tr->m_y0 + z * tr->m_ty;
    return xx*xx + yy * yy;
 }

__device__ void solve (track *tr) {
	double den = ( tr->m_sz2 * tr->m_s0 - tr->m_sz * tr->m_sz );
	if ( fabs(den) < 10e-10 ) den = 1.;
	tr->m_tx     = ( tr->m_sxz * tr->m_s0  - tr->m_sx  * tr->m_sz ) / den;
	tr->m_x0     = ( tr->m_sx  * tr->m_sz2 - tr->m_sxz * tr->m_sz ) / den;

	den = ( tr->m_uz2 * tr->m_u0 - tr->m_uz * tr->m_uz );
	if ( fabs(den) < 10e-10 ) den = 1.;
	tr->m_ty     = ( tr->m_uyz * tr->m_u0  - tr->m_uy  * tr->m_uz ) / den;
	tr->m_y0     = ( tr->m_uy  * tr->m_uz2 - tr->m_uyz * tr->m_uz ) / den;
}

__device__ void addHit ( track *tr, int offset, int *trackIds, double *inX, double *inY, double *inZ, double *inW) {

	trackIds[offset] = tr->internalId;
	tr->trackHitsNum++;

	double z = inZ[offset];
	double x = inX[offset];
	double w = inW[offset];

	tr->m_s0  += w;
	tr->m_sx  += w * x;
	tr->m_sz  += w * z;
	tr->m_sxz += w * x * z;
	tr->m_sz2 += w * z * z;

	double y = inY[offset];

	tr->m_u0  += w;
	tr->m_uy  += w * y;
	tr->m_uz  += w * z;
	tr->m_uyz += w * y * z;
	tr->m_uz2 += w * z * z;

	if( tr->trackHitsNum > 1 ) solve(tr);
}

__device__ void setTrack(track *tr, int hit0offset, int hit1offset, int *trackIds, double *inX, double *inY, double *inZ, double *inW){
	tr->m_backward = false;
	tr->trackHitsNum = 0;

	trackIds[hit0offset] = tr->internalId;
	tr->trackHitsNum++;

	double z = inZ[hit0offset];
	double x = inX[hit0offset];
	double w = inW[hit0offset];

	tr->m_s0  = w;
	tr->m_sx  = w * x;
	tr->m_sz  = w * z;
	tr->m_sxz = w * x * z;
	tr->m_sz2 = w * z * z;

	double y = inY[hit0offset];

	tr->m_u0  = w;
	tr->m_uy  = w * y;
	tr->m_uz  = w * z;
	tr->m_uyz = w * y * z;
	tr->m_uz2 = w * z * z;

	addHit (tr, hit1offset,  trackIds, inX, inY, inZ, inW);
}

__device__ inline double chi2Hit( double x, double y, double hitX, double hitY, double hitW){
	double dx = x - hitX;
	double dy = y - hitY;
	return dx * dx * (hitW) + dy * dy * (hitW);
}
__device__ inline double xAtHit(track *tr, double z )
{
	return tr->m_x0 + tr->m_tx * z;
}

__device__ inline double yAtHit( track *tr, double z  )
{
	return tr->m_y0 + tr->m_ty * z;
}

__device__ inline double chi2Track(track *tr, double hitX, double hitY, double hitZ, double hitW)
{
	return chi2Hit( xAtHit( tr, hitZ ), yAtHit(tr, hitZ ), hitX, hitY, hitW);
}

__device__ inline double chi2(track *tr, double *hitX, double *hitY, double *hitZ, double *hitW, int usedHits[])
{
	double ch = 0.0;
	int nDoF  = -4;
	for (int i =0 ; i<m_sensNum; i++){ // or just... while(i!=m_sensNum && usedHits[i] != -1) ,
		int hitNumber = usedHits[i];
		if(hitNumber== -1){
			break;
		}
		ch += chi2Track(tr, hitX[hitNumber], hitY[hitNumber], hitZ[hitNumber], hitW[hitNumber]);
		nDoF += 2;
	}
	return ch/nDoF;
}

__device__ inline bool addHitsOnSensor( double sensZ, int sensStartPost, int sensHitsNum, double xTol, double maxChi2, track *tr,
								 	    int *tracksIds, double *inX, double *inY, double *inZ, double *inW, int threadId,
								 	    int threadsNumber,int *usedHit ) {
	if (sensHitsNum == 0 ) return false;
	double xGuess = (tr->m_x0 + tr->m_tx * sensZ) - xTol - 1;
	int lastHit = sensStartPost + sensHitsNum -1;
	if((inX[(lastHit*threadsNumber)+threadId]) < xGuess) return false;
	int startHit = sensStartPost; // może lepiej bezpośrednio się do tego odnosić
	unsigned int step = sensHitsNum;
	while ( step > 2 ) {
		step = step/2;
		if (inX[((startHit+step)*threadsNumber)+threadId] < xGuess) startHit +=step;
	}
	bool added = false;
	int tmpOffset = 0;
	for(int iH = startHit; iH<=lastHit ;++iH){ ////
		tmpOffset = (iH*threadsNumber)+threadId;
		double xPred = (tr->m_x0 + tr->m_tx * inZ[tmpOffset]);
		if ( inX[tmpOffset] + xTol < xPred ) continue;
		if ( inX[tmpOffset] - xTol > xPred ) break;
		if ( chi2Track(tr, inX[tmpOffset], inY[tmpOffset], inZ[tmpOffset], inW[tmpOffset]) < maxChi2 ) {
			addHit(tr, tmpOffset,tracksIds, inX, inY, inZ, inW );
			*usedHit = tmpOffset;
			added = true;
		}
	}
	return added;
}

__device__ inline void removeHit(track *tr, int worstHitOffset, double *inX, double *inY, double *inZ, double *inW ){
	tr->trackHitsNum--;

	double z = inZ[worstHitOffset];
	double w = inW[worstHitOffset];
	double x = inX[worstHitOffset];

	tr->m_s0  -= w;
	tr->m_sx  -= w * x;
	tr->m_sz  -= w * z;
	tr->m_sxz -= w * x * z;
	tr->m_sz2 -= w * z * z;

	double y = inY[worstHitOffset];

	tr->m_u0  -= w;
	tr->m_uy  -= w * y;
	tr->m_uz  -= w * z;
	tr->m_uyz -= w * y * z;
	tr->m_uz2 -= w * z * z;

	if( tr->trackHitsNum > 1 ) solve(tr);
}

__device__ inline void removeWorstHit(track* tr, double maxChi2, int usedHits[], int *tracksIds, double *inX, double *inY, double *inZ,
									  double *inW, int *isUsed, int threadsNumber, int threadId)
{
	double topChi2 = 1.e9;
	while( topChi2 > maxChi2 ) {
	    topChi2 = 0.0;
	    int offset = 0;
	    for (int i =0 ; i<m_sensNum; i++){
	    	offset = usedHits[i];
	    	if(offset == -1){
	    		break;
	    	}
	    	double myChi2 = chi2Track(tr, inX[offset], inY[offset], inZ[offset], inW[offset] );
	    	if ( myChi2 > topChi2 ) {
	    		topChi2 = myChi2;
	    		offset = (usedHits[i]*threadsNumber)+threadId;
	    	}
	    }
	    if ( topChi2 > maxChi2 ) {
	      tracksIds[offset] = -1;
	      isUsed[offset] = 0;
	      removeHit(tr, offset, inX, inY, inZ, inW);
	    }
	}
}

__device__ inline bool all3SensorsAreDifferent(int usedHits[], int *sensorsIds) {
    if ( sensorsIds[usedHits[0]] == sensorsIds[usedHits[1]]) return false;
    if ( sensorsIds[usedHits[0]] == sensorsIds[usedHits[2]]) return false;
    if ( sensorsIds[usedHits[1]] == sensorsIds[usedHits[2]]) return false;
    return true;
}

__device__ inline int nbUnused(int usedHits[],int *isUsed) {
	int nn = 0;
    for (int i=0; i<m_sensNum; i++){
    	int hitNumber = usedHits[i];
    	if(hitNumber== -1){
    		break;
    	}
    	if ( ! isUsed[hitNumber] ) ++nn;
    }
    return nn;
}

/*
 * topHitsNum - number of hits in the event with biggest number of hits. Other events are filled with dummy hits to have the same number of hits.
 */
__global__ void searchByPair(void *data, void* resultsKernel, int topHitsNum){

	int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	track m_track = {0};			// move to shared memory
	int trackId = 0;
	int nextHitNb = 0;
	int usedHits[48] = {-1}; //numbers of used hits

	__shared__ int 		*inHitsNum;
	__shared__ int 		*inSensInfoStartPos;
	__shared__ int 		*inSensInfoHitsNum;
	__shared__ double 	*inSensInfoZ;
	__shared__ int 		*hitsIds;
	__shared__ int 		*tracksIds;
	__shared__ int 		*sensorsIds;
	__shared__ int 		*isUsed;
	__shared__ double 	*inX;
	__shared__ double 	*inY;
	__shared__ double 	*inZ;
	__shared__ double 	*inW;

	int eventsNumber = gridDim.x * blockDim.x;			//number of all the events passed to GPU
	int dataOffset =  eventsNumber * topHitsNum; 		// between different hits parameters

	if(threadId % blockDim.x == 0){	//only first thread from the block does that - maybe not the best idea ;)
		inHitsNum 				= 		(int*)data;
		inSensInfoStartPos 		= 		inHitsNum + eventsNumber;
		inSensInfoHitsNum		= 		inSensInfoStartPos + (eventsNumber * m_sensNum);
		inSensInfoZ 			= 		(double*)(inSensInfoHitsNum + (eventsNumber * m_sensNum));
		hitsIds 				= 		(int*)(inSensInfoZ + (eventsNumber * m_sensNum));
		tracksIds 				= 		hitsIds + dataOffset;
		sensorsIds				=		tracksIds + dataOffset;
		isUsed					=		sensorsIds + dataOffset;
		inX 					= 		(double*) (isUsed + dataOffset);
		inY 					= 		inX + dataOffset;
		inZ						=		inY + dataOffset;
		inW						=		inZ + dataOffset;
	}
	__syncthreads();


//	int hitsNum = inHitsNum[threadId];					//real number of hits for this event


	track *resTracks = (track*)(resultsKernel);			//tracks are stored as array of structs - should be stored as arrays of ints (coalesced access)
	sensorInfo sensor0;	//not the best solution
	sensorInfo sensor1;

	int lastSensor = m_sensNum-1;
	int firstSensor = 2;
	for ( int sens0 = lastSensor; firstSensor <= sens0; sens0 -= 1 ) {
		int sens1 = sens0 - 2;
		sensor0.startPosition = inSensInfoStartPos[threadId+(sens0*eventsNumber)];
		sensor0.hitsNum	= inSensInfoHitsNum[threadId+(sens0*eventsNumber)];
		sensor0.z = inSensInfoZ[threadId+(sens0*eventsNumber)];

		sensor1.startPosition = inSensInfoStartPos[threadId+(sens1*eventsNumber)];
		sensor1.hitsNum	= inSensInfoHitsNum[threadId+(sens1*eventsNumber)];
		sensor1.z = inSensInfoZ[threadId+(sens1*eventsNumber)];

		int hit0offset;
		int hit1offset;

		double dxMax = m_maxXSlope * fabs( sensor1.z - sensor0.z );
		double dyMax = m_maxYSlope * fabs( sensor1.z - sensor0.z );

		int first1 = sensor1.startPosition;
		for(int i0=sensor0.startPosition; i0<(sensor0.startPosition + sensor0.hitsNum );++i0){
			hit0offset = (i0*eventsNumber)+threadId;
			if(isUsed[hit0offset]){
				continue;
			}
			double x0 = inX[hit0offset];  //TODO: check if it goes to register of stays in local (global) memory
			double y0 = inY[hit0offset];
			double xMin = x0 - dxMax;
			double xMax = x0 + dxMax;
			for(int i1 = first1; i1<(sensor1.startPosition + sensor1.hitsNum ); i1++){
				memset(usedHits, -1, m_sensNum*sizeof(int));	//czy to na pewno tutaj ma być?
				hit1offset = (i1*eventsNumber)+threadId;
				double x1 = inX[hit1offset];
				if(x1<xMin){
					first1 = i1+1;
					continue;
				}
				if (x1 > xMax) break;
				if (isUsed[hit1offset]) continue;

				double y1  = inY[hit1offset];
				if ( fabs( y1 - y0 ) > dyMax ) continue;

				m_track.internalId = trackId;
				m_track.firstHit = nextHitNb;

				setTrack(&m_track, hit0offset, hit1offset, tracksIds , inX, inY, inZ, inW);
				usedHits[0] = hit0offset;   //!!!
				usedHits[1] = hit1offset;	//!!!

		        if ( sensor0.z < m_maxZForRBeamCut ) {
		          double z_beam  = zBeam(&m_track);
		          if ( z_beam > sensor0.z ) {
		            double r2Beam = r2AtZ( z_beam, &m_track );
		            if ( r2Beam > m_maxR2Beam )  continue;
		          }
		        }

		        int extraStep = 2;
		        int extraSens = sens1-extraStep;
		        int nbMissed = 0;
		        int extraHitId = 2;
		        double lastZ = sensor1.z;
		        //double lastZ = inSensInfoZ[threadId + (sens1*eventsNumber)];
		        while ( extraSens >= 0 ) {
		            double tol     =  m_extraTol;
		            double maxChi2 =  m_maxChi2ToAdd;
		            if ( inSensInfoZ[threadId + (extraSens*eventsNumber)] < lastZ - 100.0 ) {
		            	tol     = 2 * tol;
		                maxChi2 = 2 * maxChi2;
		            }
		            bool added = addHitsOnSensor(inSensInfoZ[threadId+extraSens],inSensInfoStartPos[threadId+extraSens],inSensInfoHitsNum[threadId+extraSens], tol, maxChi2,
		            							&m_track, tracksIds, inX, inY, inZ,inW, threadId, eventsNumber, &usedHits[extraHitId]);
		            if ( added ) {
		            	extraHitId++;
		            	nbMissed = 0;
		                lastZ = inSensInfoZ[threadId + extraSens];
		            } else {
		            	nbMissed += extraStep;
		                extraStep = 1;
		            }
		            if ( m_maxMissed < nbMissed ) break;
		            	extraSens -= extraStep;
		         }

		        //== Try upstream if almost forward tracks
		        if ( sensor0.z > m_maxZForRBeamCut ) {
		          extraStep = 1;
		          extraSens = sens0 + 3;
		          nbMissed = 2;
		          while ( extraSens <= lastSensor ) {
		        	  //extra = &sensors[extraSens];
		        	  int sensOffset = threadId+(extraSens*eventsNumber);
		        	  bool added = addHitsOnSensor(inSensInfoZ[sensOffset],inSensInfoStartPos[sensOffset],inSensInfoHitsNum[sensOffset], m_extraTol, m_maxChi2ToAdd,
		        	  		            		   &m_track, tracksIds, inX, inY, inZ,inW, threadId, eventsNumber, &usedHits[extraHitId]);
		            if ( added ) {
		              nbMissed = 0;
		              extraHitId++;
		            } else {
		              nbMissed += extraStep;
		            }
		            if ( m_maxMissed < nbMissed ) break;
		            extraSens += extraStep;
		          }
		        }
		        removeWorstHit(&m_track,m_maxChi2PerHit,usedHits, tracksIds, inX, inY, inZ, inW, isUsed,eventsNumber,threadId );
		        if ( m_track.trackHitsNum < 3 ) continue;
		        //== Add compatible hits in sens0 and sens1.
		        int tmpOffset;
		        if(i0 != (sensor0.startPosition + sensor0.hitsNum -1)){
		        	tmpOffset = ((i0+1)*eventsNumber)+threadId ;
		        	if ( chi2Track(&m_track, inX[tmpOffset], inY[tmpOffset],inZ[tmpOffset],inW[tmpOffset]) < m_maxChi2SameSensor) {
		        		++i0;
		                addHit(&m_track, hit0offset, tracksIds, inX, inY, inZ, inW );
		            }
		        }
		        if (i1 != (sensor1.startPosition + sensor0.hitsNum -1) ) {
		        	tmpOffset = ((i1+1)*eventsNumber)+threadId ;
		        	if ( chi2Track(&m_track, inX[tmpOffset], inY[tmpOffset],inZ[tmpOffset],inW[tmpOffset]) < m_maxChi2SameSensor) {
		        		++i1;
		        		addHit(&m_track, hit1offset, tracksIds, inX, inY, inZ, inW );
		            }
		        }
		        //== Final check: if only 3 hits, all should be unused and chi2 good.
		        if ( m_track.trackHitsNum == 3 ) {
		        	if ( !all3SensorsAreDifferent(usedHits, sensorsIds) ) {
		                continue;
		            }
		        	if(nbUnused(usedHits,isUsed) != 3){
		        		continue;
		            }
		            if(chi2(&m_track, inX, inY, inZ, inW, usedHits) > m_maxChi2Short){
		            	continue;
		            }
		        } else {
		        	if ( nbUnused(usedHits,isUsed) < .6 * m_track.trackHitsNum ) {
		        		continue;
		        	}
		        }
		        resTracks[trackId] = m_track; ///--- numer ścieżki

				trackId++;
				nextHitNb += m_track.trackHitsNum;

				if ( m_track.trackHitsNum > 3 ) {
					for (int i =0 ; i<48; i++){
						int hitNumber = usedHits[i];
						if(hitNumber== -1){
							break;
					 	}
						isUsed[hitNumber] = 1;
				    }
					break;
				}
			} //i1
		} //i0
	} //sensor0
//	resStats[0] = trackId;
}

inline int _ConvertSMVer2Cores(int major, int minor)
{
    // defines for GPU Architecture types
    typedef struct
    {
        int SM; // 0xMm, M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
        { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
        { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
        { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
        { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
        { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
    };

    int index = 0;
    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }
        index++;
    }
    return nGpuArchCoresPerSM[7].Cores;
}

inline int getBestDevice()
{
    int current_device     = 0, sm_per_multiproc  = 0;
    int max_compute_perf   = 0, max_perf_device   = 0;
    int device_count       = 0, best_SM_arch      = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount(&device_count);

    // Find the best major SM Architecture GPU device
    while (current_device < device_count)
    {
        hipGetDeviceProperties(&deviceProp, current_device);
        if (deviceProp.computeMode != hipComputeModeProhibited)
        {
            if (deviceProp.major > 0 && deviceProp.major < 9999)
            {
                best_SM_arch = MAX(best_SM_arch, deviceProp.major);
            }
        }
        current_device++;
    }
    // Find the best CUDA capable GPU device
    current_device = 0;
    while (current_device < device_count)
    {
        hipGetDeviceProperties(&deviceProp, current_device);
        // If this GPU is not running on Compute Mode prohibited, then we can add it to the list
        if (deviceProp.computeMode != hipComputeModeProhibited)
        {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999)
            {
                sm_per_multiproc = 1;
            }
            else
            {
                sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
            }

            int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;

            if (compute_perf  > max_compute_perf)
            {
                // If we find GPU with SM major > 2, search only these
                if (best_SM_arch > 2)
                {
                    // If our device==dest_SM_arch, choose this, or else pass
                    if (deviceProp.major == best_SM_arch)
                    {
                        max_compute_perf  = compute_perf;
                        max_perf_device   = current_device;
                    }
                }
                else
                {
                    max_compute_perf  = compute_perf;
                    max_perf_device   = current_device;
                }
            }
        }
        ++current_device;
    }
    return max_perf_device;
}

void findCudaDevice()
{
    hipDeviceProp_t deviceProp;
    int devID = 0;
    devID = getBestDevice();
    CUDA_CHECK_RETURN(hipSetDevice(devID));
    CUDA_CHECK_RETURN(hipGetDeviceProperties(&deviceProp, devID));
    printf("> Using CUDA device [%d]: %s\n", devID, deviceProp.name);
}

/*
 * Function that starts GPU Kernel.
 */
void launchKernel(vector<char> &inputVector, int blocksPerGrid, int threadsPerBlock, unsigned int allHits, int topHitsNum, char **results){
	findCudaDevice(); //choosing the fastest (GFLOPS) card on the system
	void *vecKernel = NULL;
	void *resultsKernel = NULL;

	CUDA_CHECK_RETURN(hipDeviceReset());
	//hipDeviceSetCacheConfig(hipFuncCachePreferL1); //TODO: we can increase L1 cache if we don't use shared mem
	int resultsInBytes = (allHits)*(sizeof(track) + 2*sizeof(int));   //Allocating memory for as many tracks as hits...we can allocate 1/3 of it but then we must know all the offeset between events
	CUDA_CHECK_RETURN(hipMalloc((void**) &vecKernel, inputVector.size()));
	CUDA_CHECK_RETURN(hipMemcpy(vecKernel, &inputVector[0], inputVector.size(), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMalloc((void**) &resultsKernel, resultsInBytes));

	searchByPair<<<blocksPerGrid, threadsPerBlock>>>(vecKernel,resultsKernel,topHitsNum);
//	searchByPair<<<1,4>>>(vecKernel,resultsKernel, allHits); //testing

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());

	*results = (char*)malloc(resultsInBytes);  //TODO: release 'results' memory
	CUDA_CHECK_RETURN(hipMemcpy(*results, resultsKernel, resultsInBytes, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree((void*) vecKernel));
	CUDA_CHECK_RETURN(hipFree((void*) resultsKernel));
	CUDA_CHECK_RETURN(hipDeviceReset());

}
